#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>

inline hipError_t checkCudaErrors(hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Runtime error: %s\n", hipGetErrorString(err));
  }
  return err;
}

cv::Mat imageInputRGBA;
cv::Mat imageOutputRGBA;

uchar4 *d_inputImageRGBA__;
uchar4 *d_outputImageRGBA__;

float *h_filter__;

size_t numRows() { return imageInputRGBA.rows; }
size_t numCols() { return imageInputRGBA.cols; }

void preProcess(uchar4 **h_inputImageRGBA, uchar4 **h_outputImageRGBA,
                uchar4 **d_inputImageRGBA, uchar4 **d_outputImageRGBA,
                unsigned char **d_redBlurred,
                unsigned char **d_greenBlurred,
                unsigned char **d_blueBlurred,
                float **h_filter, int *filterWidth,
                const std::string &filename) {

  //make sure the context initializes ok
  checkCudaErrors(hipFree(0));

  cv::Mat image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
  if (image.empty()) {
    std::cerr << "Couldn't open file: " << filename << std::endl;
    exit(1);
  }

  cv::cvtColor(image, imageInputRGBA, CV_BGR2RGBA);

  //allocate memory for the output
  imageOutputRGBA.create(image.rows, image.cols, CV_8UC4);

  //This shouldn't ever happen given the way the images are created
  //at least based upon my limited understanding of OpenCV, but better to check
  if (!imageInputRGBA.isContinuous() || !imageOutputRGBA.isContinuous()) {
    std::cerr << "Images aren't continuous!! Exiting." << std::endl;
    exit(1);
  }

  *h_inputImageRGBA  = (uchar4 *)imageInputRGBA.ptr<unsigned char>(0);
  *h_outputImageRGBA = (uchar4 *)imageOutputRGBA.ptr<unsigned char>(0);

  const size_t numPixels = numRows() * numCols();
  //allocate memory on the device for both input and output
  checkCudaErrors(hipMalloc(d_inputImageRGBA, sizeof(uchar4) * numPixels));
  checkCudaErrors(hipMalloc(d_outputImageRGBA, sizeof(uchar4) * numPixels));
  checkCudaErrors(hipMemset(*d_outputImageRGBA, 0, numPixels * sizeof(uchar4))); //make sure no memory is left laying around

  //copy input array to the GPU
  checkCudaErrors(hipMemcpy(*d_inputImageRGBA, *h_inputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));

  d_inputImageRGBA__  = *d_inputImageRGBA;
  d_outputImageRGBA__ = *d_outputImageRGBA;

  //now create the filter that they will use
  const int blurKernelWidth = 9;
  const float blurKernelSigma = 2.;

  *filterWidth = blurKernelWidth;

  //create and fill the filter we will convolve with
  *h_filter = new float[blurKernelWidth * blurKernelWidth];
  h_filter__ = *h_filter;

  float filterSum = 0.f; //for normalization

  for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) {
    for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) {
      float filterValue = expf( -(float)(c * c + r * r) / (2.f * blurKernelSigma * blurKernelSigma));
      (*h_filter)[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] = filterValue;
      filterSum += filterValue;
    }
  }

  float normalizationFactor = 1.f / filterSum;

  for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) {
    for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) {
      (*h_filter)[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] *= normalizationFactor;
    }
  }

  //blurred
  checkCudaErrors(hipMalloc(d_redBlurred,    sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMalloc(d_greenBlurred,  sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMalloc(d_blueBlurred,   sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_redBlurred,   0, sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_greenBlurred, 0, sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_blueBlurred,  0, sizeof(unsigned char) * numPixels));
}

void postProcess(const std::string& output_file) {
  const int numPixels = numRows() * numCols();
  //copy the output back to thehost
  checkCudaErrors(hipMemcpy(imageOutputRGBA.ptr<unsigned char>(0), d_outputImageRGBA__, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost));

  cv::Mat imageOutputBGR;
  cv::cvtColor(imageOutputRGBA, imageOutputBGR, CV_RGBA2BGR);
  //output the image
  cv::imwrite(output_file.c_str(), imageOutputBGR);

  //cleanup
  hipFree(d_inputImageRGBA__);
  hipFree(d_outputImageRGBA__);
  delete[] h_filter__;
}




__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  // TODO
  
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  // TODO
  //
  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //TODO:
  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc

  //TODO:
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!

}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  //TODO: Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize;

  //TODO:
  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the image size and and block size.
  const dim3 gridSize;

  //TODO: Launch a kernel for separating the RGBA image into different color channels

  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  //TODO: Call your convolution kernel here 3 times, once for each color channel.

  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Now we recombine your results. We take care of launching this kernel for you.
  //
  // NOTE: This kernel launch depends on the gridSize and blockSize variables,
  // which you must set yourself.
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
}



int main(int argc, char **argv) {
  uchar4 *h_inputImageRGBA,  *d_inputImageRGBA;
  uchar4 *h_outputImageRGBA, *d_outputImageRGBA;
  unsigned char *d_redBlurred, *d_greenBlurred, *d_blueBlurred;

  float *h_filter;
  int    filterWidth;

  std::string input_file;
  std::string output_file;
  if (argc == 3) {
    input_file  = std::string(argv[1]);
    output_file = std::string(argv[2]);
  }
  else {
    std::cerr << "Usage: ./hw input_file output_file" << std::endl;
    exit(1);
  }
  //load the image and give us our input and output pointers
  preProcess(&h_inputImageRGBA, &h_outputImageRGBA, &d_inputImageRGBA, &d_outputImageRGBA,
             &d_redBlurred, &d_greenBlurred, &d_blueBlurred,
             &h_filter, &filterWidth, input_file);

  allocateMemoryAndCopyToGPU(numRows(), numCols(), h_filter, filterWidth);
  //GpuTimer timer;
  //timer.Start();
  //call the students' code
  your_gaussian_blur(h_inputImageRGBA, d_inputImageRGBA, d_outputImageRGBA, numRows(), numCols(),
                     d_redBlurred, d_greenBlurred, d_blueBlurred, filterWidth);
  //timer.Stop();
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  //int err = printf("%f msecs.\n", timer.Elapsed());
  int err = -1;

  if (err < 0) {
    //Couldn't print! Probably the student closed stdout - bad news
    std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
    exit(1);
  }

  cleanup();
  //check results and output the blurred image
  postProcess(output_file);

  checkCudaErrors(hipFree(d_redBlurred));
  checkCudaErrors(hipFree(d_greenBlurred));
  checkCudaErrors(hipFree(d_blueBlurred));

  return 0;
} 
