#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
__global__ void add(int *a, int *b, int *c) {
*c = *a + *b;
}

__global__ void cube(float * d_out, float * d_in){
	// Todo: Fill in this function
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f * f;
}

int main() {
  const int ARRAY_SIZE = 94;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

  float *d_in;
  float *d_out;

  hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	cube<<<1, ARRAY_SIZE>>>(d_out, d_in);
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  for (int i =0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

  hipFree(d_in);
	hipFree(d_out);

  int a, b, c;
  // host copies of variables a, b & c
  int *d_a, *d_b, *d_c;
  
  // device copies of variables a, b & c
  int size = sizeof(int);
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Setup input values  
  c = 0;
  a = 3;
  b = 5;
  // Copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU
  add<<<1,1>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    if(err!=hipSuccess) {
        printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
    }
  printf("results is %d\n",c);
  // Cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}